#include "hip/hip_runtime.h"
/*
Copyright (c) 2011-2013 Gerhard Reitmayr, TU Graz

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "kfusion.h"
#include "perfstats.h"
#include "track.h"
//#include "helpers.h"

#include <thrust/device_vector.h>

#include <iostream>
#include <GL/glut.h>
#include <GL/gl.h>

using namespace std;

PerfStats Stats;

__global__ void setSphere( Volume volume, const float3 center, const float radius, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float d = length(volume.pos(pos) - center);
        if(d < radius)
            volume.set(pos, make_float2(val, 0.0f));
    }
}

__global__ void setBox( Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float3 p = volume.pos(pos);
        if(min_corner.x < p.x && min_corner.y < p.y && min_corner.z < p.z && 
           p.x < max_corner.x && p.y < max_corner.y && p.z < max_corner.z )
            volume.set(pos, make_float2(val, 0.0f));
    }
}

void initVolumeWrap( Volume volume, const float val ){
    dim3 block(32,16);
    initVolume<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, make_float2(val, 0.0f));
}

void setBoxWrap(Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    dim3 block(32,16);
    setBox<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, min_corner, max_corner, val);
}

void setSphereWrap(Volume volume, const float3 center, const float radius, const float val ){
    dim3 block(32,16);
    setSphere<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, center, radius, val);
}

__global__ void renderNormals( Image<uchar3> out, const Image<float3> in ){
    float3 n = in.el();
    if(n.x == -2)
        out.el() = make_uchar3(0,0,0);
    else {
        n = normalize(n);
        out.el() = make_uchar3(n.x*128 + 128, n.y*128+128, n.z*128+128);
    }
}

void renderNormalMap( Image<uchar3> out, const Image<float3> & normal ){
    dim3 block(20,20);
    renderNormals<<<divup(normal.size, block), block>>>( out, normal );
}

__global__ void renderLightKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const float3 light, const float3 ambient ){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f);
        const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
        out.el() = make_uchar4(col.x, col.y, col.z, 255);
    }
}

void renderLight( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const float3 light, const float3 ambient ){
    dim3 block(32,16);
    renderLightKernel<<<divup(out.size, block), block>>>( out, vertex, normal, light, ambient );
}

__global__ void renderTextureKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const Image<uchar3> texture, const Matrix4 texproj, const float3 light){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * vertex.el();
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderTexture( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const Image<uchar3> & texture, const Matrix4 & texproj, const float3 light){
    dim3 block(32,16);
    renderTextureKernel<<<divup(out.size, block), block>>>( out, vertex, normal, texture, texproj, light);
}

__global__ void renderDepth( Image<uchar3> out, const Image<float> depth, const float nearPlane, const float farPlane){
    const float d = (clamp(depth.el(), nearPlane, farPlane) - nearPlane) / (farPlane - nearPlane);
    out.el() = make_uchar3(d * 255, d * 255, d * 255);
}

void renderDepthMap( Image<uchar3> out, const Image<float> & depth, const float nearPlane, const float farPlane ){
    dim3 block(32,16);
    renderDepth<<<divup(depth.size, block), block>>>( out, depth, nearPlane, farPlane );
}

__global__ void renderTrack( Image<uchar4> out, const Image<TrackData> data ){
    const uint2 pos = thr2pos2();
    switch(data[pos].result){
    case 1: out[pos] = make_uchar4(128, 128, 128,0);  // ok
        break;
    case -1: out[pos] = make_uchar4(0, 0, 0,0);      // no input
        break;
    case -2: out[pos] = make_uchar4(255,0,0,0);        // not in image
        break;
    case -3:  out[pos] = make_uchar4(0,255,0,0);        // no correspondence
        break;
    case -4: out[pos] = make_uchar4(0,0,255,0);        // to far away
        break;
    case -5: out[pos] = make_uchar4(255,255,0,0);     // wrong normal
        break;
    }
}

void renderTrackResult( Image<uchar4> out, const Image<TrackData> & data ){
    dim3 block(32,16);
    renderTrack<<<divup(out.size, block), block>>>( out, data );
}

__global__ void raycastLight( Image<uchar4> render, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    const uint2 pos = thr2pos2();
    int2 posS = make_int2(pos.x, pos.y);
    
    float4 hit = raycast( volume, posS, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        const float3 test = make_float3(hit);
        const float3 surfNorm = volume.grad(test);
        if(length(surfNorm) > 0){
            const float3 diff = normalize(light - test);
            const float dir = fmaxf(dot(normalize(surfNorm), diff), 0.f);
            const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
            render.el() = make_uchar4(col.x, col.y, col.z,0);
        } else {
            render.el() = make_uchar4(0,0,0,0);
        }
    } else {
        render.el() = make_uchar4(0,0,0,0);
    }
}

void renderVolumeLight( Image<uchar4> out, const Volume & volume, const Matrix4 view, const float nearPlane, const float farPlane, const float largestep, const float3 light, const float3 ambient ){
    dim3 block(16,16);
    raycastLight<<<divup(out.size, block), block>>>( out,  volume, view, nearPlane, farPlane, volume.dim.x/volume.size.x, largestep, light, ambient );
}

__global__ void raycastInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    const uint2 pos = thr2pos2();

    int2 transPos = make_int2(pos.x, pos.y);
    transPos.x -= (outputSize.x - 640) / 2;
    transPos.y -= (outputSize.y - 480) / 2;
    
    float4 hit = raycast( volume, transPos, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        pos3D[pos] = make_float3(hit);
        depth[pos] = hit.w;
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal[pos].x = -2;
        } else {
            normal[pos] = normalize(surfNorm);
        }
    } else {
        pos3D[pos] = make_float3(0);
        normal[pos] = make_float3(0);
        depth[pos] = 0;
    }
}

void renderInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    dim3 block(32,16);
    raycastInput<<<divup(pos3D.size, block), block>>>(pos3D, normal, depth, volume, view, nearPlane, farPlane, step, largestep, outputSize);
}

__global__ void OculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const Image<bool> gridWroteOn) {
    float f = 204.4f;
    float2 iResolution = make_float2(1280, 800);
    float DistortionScale = 1.71461;
    float pp_adjust = 48.62;
    float3 K = make_float3(1.00f, 0.22f, 0.24f);

    int resX = 1000; // you have to update the ones in kinect.cpp
    int resY = 1000;
    int resZ = 1000;

    const uint2 pos = thr2pos2();
    float2 uv = make_float2(pos.x, pos.y);
    
    // Which eye?
    float i = uv.x <= iResolution.x/2.0 ? 0.0 : 1.0;

    // Compute Principle point
    float sx = i*2.0 - 1.0;
    float2 pp = make_float2( sx*(iResolution.x / 4.0 - pp_adjust) + iResolution.x / 2.0, iResolution.y / 2.0 );

    // Distort uv for Oculus (using res independant coords)
    float2 theta = make_float2(4.0 * (uv.x-pp.x) / iResolution.x, 4.0 * (uv.y-pp.y) / iResolution.x);
    float rSq= theta.x * theta.x + theta.y * theta.y;
    uv.x = pp.x + iResolution.x * theta.x * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);
    uv.y = pp.y + iResolution.x * theta.y * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);

    float2 uvMinusPp = make_float2(uv.x - pp.x, uv.y - pp.y);
    uvMinusPp.x /= f;
    uvMinusPp.y /= f;

    float3 ray = make_float3(uvMinusPp.x, uvMinusPp.y, 1.0f);
    float rayNorm = sqrt(ray.x*ray.x + ray.y*ray.y + ray.z*ray.z);
    ray.x /= rayNorm;
    ray.y /= rayNorm;
    ray.z /= rayNorm;

    float4 hit;
    float3 pos3D;
    float3 normal;

    hit = raycastDirPos(volume, view, nearPlane, farPlane, step, largestep, ray);

    if(hit.w > 0){
        pos3D = make_float3(hit);
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal.x = -2;
        } else {
            normal = normalize(surfNorm);
        }
    } else {
        pos3D = make_float3(0);
        normal = make_float3(0);
    }

    if(normal.x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * pos3D;
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - pos3D);
        const float dir = fmaxf(dot(normal, diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            int coordX = pos3D.x*resX/5;
            int coordY = pos3D.y*resY/5;
            int coordZ = pos3D.z*resZ/5;
            bool wroteOn = false;
            if(0 <= coordX && coordX < resX && 0 <= coordY && coordY < resY && 0 <= coordZ && coordZ < resZ) {
                wroteOn = gridWroteOn[make_uint2(coordX+resX*coordY+resX*resY*coordZ,0)];
            }
            if(wroteOn)
                out.el() = make_uchar4(texcol.x*dir/3.0f, texcol.y*dir/3.0f, 255*dir, 255);
            else
                out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderOculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> & texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const Image<bool> & gridWroteOn){
    dim3 block(32,16);
    OculusCam<<<divup(out.size, block), block>>>(out, volume, texture, view, texproj, nearPlane, farPlane, step, largestep, light, gridWroteOn);
}

void viewMatrixUpdate(Matrix4 & ovrPose, float yYaw, float zEyeRoll, float xEyePitch){
    ovrPose.data[0].x = cos(yYaw)*cos(zEyeRoll) + sin(yYaw)*sin(xEyePitch)*sin(zEyeRoll);
    ovrPose.data[0].y = cos(zEyeRoll)*sin(yYaw)*sin(xEyePitch) - cos(yYaw)*sin(zEyeRoll);
    ovrPose.data[0].z = cos(xEyePitch)*sin(yYaw);
        
    ovrPose.data[1].x = cos(xEyePitch)*sin(zEyeRoll);
    ovrPose.data[1].y = cos(xEyePitch)*cos(zEyeRoll);
    ovrPose.data[1].z = - sin(xEyePitch);
        
    ovrPose.data[2].x = cos(yYaw)*sin(xEyePitch)*sin(zEyeRoll) - cos(zEyeRoll)*sin(yYaw);
    ovrPose.data[2].y = sin(yYaw)*sin(zEyeRoll) + cos(yYaw)*cos(zEyeRoll)*sin(xEyePitch);
    ovrPose.data[2].z = cos(yYaw)*cos(xEyePitch);
}

__global__ void isSquare(Image<uchar3> texture){
    const uint2 pos = thr2pos2();
    if ((pos.x == 310 && pos.y < 250 && pos.y > 230) || (pos.x == 330 && pos.y < 250 && pos.y > 230) || (pos.y == 230 && pos.x > 310 && pos.x < 330) || (pos.y == 250 && pos.x > 310 && pos.x < 330)) 
        texture.el() =  make_uchar3(255,0,0);

}

void drawSquare(Image<uchar3> & texture){
    dim3 block(32,16);
    isSquare<<<divup(texture.size, block), block>>>(texture);
}

__global__ void trim(Image<uchar3> texture, const float3 hsvToTrack) {
    uchar3 pixColor = texture.el();
    const uint2 pos = thr2pos2();

    float r = ((float)pixColor.x) / 255.0f;
    float g = ((float)pixColor.y) / 255.0f;
    float b = ((float)pixColor.z) / 255.0f;

    float epsiHue = 0.2f;

    float hue, sat;
    float max = (r > g) ? (r > b) ? r : b : (g > b) ? g : b;
    float min = (r < g) ? (r < b) ? r : b : (g < b) ? g : b;

    if (max == min) {
        hue = 0;
    } else if (max == r) {
        hue = (60.0f*(g-b)/(max-min) + 360.0f);
        if (hue > 360.0f) {
            hue -= 360.0f;
        }
    } else if (max == g){
                hue = (60.0f*(b-r)/(max-min) + 120.0f);
    } else {
        hue = (60.0f*(r-g)/(max-min) + 240.0f);
    }

    hue = hue*3.14159/360.0f;

    if (max == 0)
        sat = 0;
    else
    sat = 1.0f - min/max;

    float diffSat, diffHue;
    diffSat = (sat > hsvToTrack.y) ? sat - hsvToTrack.y : hsvToTrack.y - sat;
    diffHue = (hue + 3.14159 - hsvToTrack.x);
    if (diffHue > 3.14159)
        diffHue -= 3.14159;

    if (diffHue < epsiHue && diffSat < hsvToTrack.y/4.0f) {
        texture.el() = make_uchar3(255,0,0);
    }
}

void colorTrim(Image<uchar3> & texture, const float3 hsvToTrack) {
    dim3 block(32,16);
    trim<<<divup(texture.size, block), block>>>(texture, hsvToTrack);
}

__host__ __device__ float3 huesatval(const Image<uchar3> & texture, uint2 pos){
    uchar3 pixColor = texture[pos];
    float r = ((float)pixColor.x) / 255.0f;
    float g = ((float)pixColor.y) / 255.0f;
    float b = ((float)pixColor.z) / 255.0f;

    float hue, sat, val;
    float max = (r > g) ? (r > b) ? r : b : (g > b) ? g : b;
    float min = (r < g) ? (r < b) ? r : b : (g < b) ? g : b;

    if (max == min) {
        hue = 0;
    } else if (max == r) {
        hue = (60.0f*(g-b)/(max-min) + 360.0f);
        if (hue > 360.0f) {
            hue -= 360.0f;
        }
    } else if (max == g){
        hue = (60.0f*(b-r)/(max-min) + 120.0f);
    } else {
        hue = (60.0f*(r-g)/(max-min) + 240.0f);
    }

    hue = hue*3.14159/360.0f;

    if (max == 0)
        sat = 0;
    else
        sat = 1.0f - min/max;

    val = max;

    return make_float3(hue, sat, val);
}

// __global__ void mapGPU(const Image<uchar3> texture, char* mapPix, const float3 hsvToTrack) {
//     uint2 pos = thr2pos2();
//     if (pos.x > 319) {
//         pos.x = 0;
//     }
//     if (pos.y > 239) {
//         pos.y = 0;
//     }

//     float3 p1 = huesatval(texture, make_uint2(pos.x,pos.y));
//     float3 p2 = huesatval(texture, make_uint2(pos.x+1,pos.y));
//     float3 p3 = huesatval(texture, make_uint2(pos.x,pos.y+1));
//     float3 p4 = huesatval(texture, make_uint2(pos.x+1,pos.y));

//     __shared__ char* testMap;
//     testMap = mapPix;

//     float epsiHue = .2f;
//     *(testMap + pos.x*240 + pos.y) = 0;
//     float diffSat1, diffHue1, diffSat2, diffHue2, diffSat3, diffHue3, diffSat4, diffHue4;
    
//     diffSat1 = (p1.y > hsvToTrack.y) ? p1.y - hsvToTrack.y : hsvToTrack.y - p1.y;
//     diffHue1 = (p1.x + 3.14159 - hsvToTrack.x);
//     if (diffHue1 > 3.14159)
//         diffHue1 -= 3.14159;
//     if (diffHue1 < epsiHue && diffSat1 < hsvToTrack.y/4.0f)
//         //*(mapPix + pos.x*240 + pos.y) += 1;

//     diffSat2 = (p2.y > hsvToTrack.y) ? p2.y - hsvToTrack.y : hsvToTrack.y - p2.y;
//     diffHue2 = (p2.x + 3.14159 - hsvToTrack.x);
//     if (diffHue2 > 3.14159)
//         diffHue2 -= 3.14159;
//     if (diffHue2 < epsiHue && diffSat2 < hsvToTrack.y/4.0f)
//         //*(mapPix + pos.x*240 + pos.y) += 1;

//     diffSat3 = (p3.y > hsvToTrack.y) ? p3.y - hsvToTrack.y : hsvToTrack.y - p3.y;
//     diffHue3 = (p3.x + 3.14159 - hsvToTrack.x);
//     if (diffHue3 > 3.14159)
//         diffHue3 -= 3.14159;
//     if (diffHue3 < epsiHue && diffSat3 < hsvToTrack.y/4.0f)
//         //*(mapPix + pos.x*240 + pos.y) += 1;

//     diffSat4 = (p4.y > hsvToTrack.y) ? p4.y - hsvToTrack.y : hsvToTrack.y - p4.y;
//     diffHue4 = (p4.x + 3.14159 - hsvToTrack.x);
//     if (diffHue4 > 3.14159)
//         diffHue4 -= 3.14159;
//     //if (diffHue4 < epsiHue && diffSat4 < hsvToTrack.y/4.0f)
//         //*(mapPix + pos.x*240 + pos.y) += 1;
    
// }

// void computeMapGPU(const Image<uchar3> & texture, char* mapPix, const float3 hsvToTrack) {
//     dim3 block(32,16);
//     uint2 size = make_uint2(320,240);
//     mapGPU<<<divup(size, block), block>>>(texture, mapPix, hsvToTrack);
// }

// __global__ void heatmapGPU(char* mapPix, int* heatMap, int radius) {
//     const uint2 pos = thr2pos2();
//     *(heatMap + pos.x * 240 + pos.y) = 0;
//     for(int k = pos.x-radius; k < pos.x+radius+1; k++) {
//         for(int l = pos.y-radius; l < pos.y+radius+1; l++) {
//             if (k>=0 && k<320 && l>=0 && l<240)
//                 *(heatMap + pos.x * 240 + pos.y) += *(mapPix + k * 240 + l);
//         }
//     }
// }

// void computeHeatMapGPU(char* mapPix, int* heatMap, int radius) {
//     dim3 block(32,16);
//     uint2 size = make_uint2(320,240);
//     heatmapGPU<<<divup(size, block), block>>>(mapPix, heatMap, radius);
// }