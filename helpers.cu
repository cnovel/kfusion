#include "hip/hip_runtime.h"
/*
Copyright (c) 2011-2013 Gerhard Reitmayr, TU Graz

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "kfusion.h"
#include "perfstats.h"
#include <iostream>
#include <GL/glut.h>
#include <GL/gl.h>

using namespace std;

PerfStats Stats;

__global__ void setSphere( Volume volume, const float3 center, const float radius, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float d = length(volume.pos(pos) - center);
        if(d < radius)
            volume.set(pos, make_float2(val, 0.0f));
    }
}

__global__ void setBox( Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float3 p = volume.pos(pos);
        if(min_corner.x < p.x && min_corner.y < p.y && min_corner.z < p.z && 
           p.x < max_corner.x && p.y < max_corner.y && p.z < max_corner.z )
            volume.set(pos, make_float2(val, 0.0f));
    }
}

void initVolumeWrap( Volume volume, const float val ){
    dim3 block(32,16);
    initVolume<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, make_float2(val, 0.0f));
}

void setBoxWrap(Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    dim3 block(32,16);
    setBox<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, min_corner, max_corner, val);
}

void setSphereWrap(Volume volume, const float3 center, const float radius, const float val ){
    dim3 block(32,16);
    setSphere<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, center, radius, val);
}

__global__ void renderNormals( Image<uchar3> out, const Image<float3> in ){
    float3 n = in.el();
    if(n.x == -2)
        out.el() = make_uchar3(0,0,0);
    else {
        n = normalize(n);
        out.el() = make_uchar3(n.x*128 + 128, n.y*128+128, n.z*128+128);
    }
}

void renderNormalMap( Image<uchar3> out, const Image<float3> & normal ){
    dim3 block(20,20);
    renderNormals<<<divup(normal.size, block), block>>>( out, normal );
}

__global__ void renderLightKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const float3 light, const float3 ambient ){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f);
        const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
        out.el() = make_uchar4(col.x, col.y, col.z, 255);
    }
}

void renderLight( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const float3 light, const float3 ambient ){
    dim3 block(32,16);
    renderLightKernel<<<divup(out.size, block), block>>>( out, vertex, normal, light, ambient );
}

__global__ void renderTextureKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const Image<uchar3> texture, const Matrix4 texproj, const float3 light){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * vertex.el();
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderTexture( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const Image<uchar3> & texture, const Matrix4 & texproj, const float3 light){
    dim3 block(32,16);
    renderTextureKernel<<<divup(out.size, block), block>>>( out, vertex, normal, texture, texproj, light);
}

__global__ void renderDepth( Image<uchar3> out, const Image<float> depth, const float nearPlane, const float farPlane){
    const float d = (clamp(depth.el(), nearPlane, farPlane) - nearPlane) / (farPlane - nearPlane);
    out.el() = make_uchar3(d * 255, d * 255, d * 255);
}

void renderDepthMap( Image<uchar3> out, const Image<float> & depth, const float nearPlane, const float farPlane ){
    dim3 block(32,16);
    renderDepth<<<divup(depth.size, block), block>>>( out, depth, nearPlane, farPlane );
}

__global__ void renderTrack( Image<uchar4> out, const Image<TrackData> data ){
    const uint2 pos = thr2pos2();
    switch(data[pos].result){
    case 1: out[pos] = make_uchar4(128, 128, 128,0);  // ok
        break;
    case -1: out[pos] = make_uchar4(0, 0, 0,0);      // no input
        break;
    case -2: out[pos] = make_uchar4(255,0,0,0);        // not in image
        break;
    case -3:  out[pos] = make_uchar4(0,255,0,0);        // no correspondence
        break;
    case -4: out[pos] = make_uchar4(0,0,255,0);        // to far away
        break;
    case -5: out[pos] = make_uchar4(255,255,0,0);     // wrong normal
        break;
    }
}

void renderTrackResult( Image<uchar4> out, const Image<TrackData> & data ){
    dim3 block(32,16);
    renderTrack<<<divup(out.size, block), block>>>( out, data );
}

__global__ void raycastLight( Image<uchar4> render, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    const uint2 pos = thr2pos2();
    int2 posS = make_int2(pos.x, pos.y);
    
    float4 hit = raycast( volume, posS, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        const float3 test = make_float3(hit);
        const float3 surfNorm = volume.grad(test);
        if(length(surfNorm) > 0){
            const float3 diff = normalize(light - test);
            const float dir = fmaxf(dot(normalize(surfNorm), diff), 0.f);
            const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
            render.el() = make_uchar4(col.x, col.y, col.z,0);
        } else {
            render.el() = make_uchar4(0,0,0,0);
        }
    } else {
        render.el() = make_uchar4(0,0,0,0);
    }
}

void renderVolumeLight( Image<uchar4> out, const Volume & volume, const Matrix4 view, const float nearPlane, const float farPlane, const float largestep, const float3 light, const float3 ambient ){
    dim3 block(16,16);
    raycastLight<<<divup(out.size, block), block>>>( out,  volume, view, nearPlane, farPlane, volume.dim.x/volume.size.x, largestep, light, ambient );
}

__global__ void raycastInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    const uint2 pos = thr2pos2();

    int2 transPos = make_int2(pos.x, pos.y);
    transPos.x -= (outputSize.x - 640) / 2;
    transPos.y -= (outputSize.y - 480) / 2;
    
    float4 hit = raycast( volume, transPos, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        pos3D[pos] = make_float3(hit);
        depth[pos] = hit.w;
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal[pos].x = -2;
        } else {
            normal[pos] = normalize(surfNorm);
        }
    } else {
        pos3D[pos] = make_float3(0);
        normal[pos] = make_float3(0);
        depth[pos] = 0;
    }
}


void renderInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    dim3 block(32,16);
    raycastInput<<<divup(pos3D.size, block), block>>>(pos3D, normal, depth, volume, view, nearPlane, farPlane, step, largestep, outputSize);
}

__global__ void hmd_barrel_bis(Image<uchar4> out, const Image<uchar4> viewLeft, const Image<uchar4> viewRight) {
    int h = 800;
    int w = 1280;

    float hmd_zoom = 0.95f;
    int half = w/2;

    float2 hmd_scale_out = make_float2(hmd_zoom * half, hmd_zoom * h);
    float2 hmd_scale_in = make_float2(1.0f/half, 1.0f/h);
    float4 hmd_warp_param = make_float4(1.00f, 0.22f, 0.24f, 0.00f); // these should come from the OVR SDK

    const uint2 pos = thr2pos2();
    int2 posS = make_int2(pos.x, pos.y);

    if (posS.x < half) {
        int2 hmd_lens_center = make_int2(half/2, h/2);
        const float2    v   = make_float2(hmd_scale_in.x * (posS.x - hmd_lens_center.x), hmd_scale_in.y * (posS.y - hmd_lens_center.y));
        const float     rr  = v.x*v.x + v.y*v.y;
        const float4    r   = make_float4(1, rr, rr*rr, rr*rr*rr);
        const float     product = hmd_warp_param.x*r.x + hmd_warp_param.y*r.y + hmd_warp_param.z*r.z + hmd_warp_param.w*r.w;
        const float2    vOut = make_float2(hmd_scale_out.x * v.x * product + hmd_lens_center.x, hmd_scale_out.y * v.y * product + hmd_lens_center.y);
        const int2      posOutS = make_int2(int(vOut.x), int(vOut.y));
        if (posOutS.x >= 0 && posOutS.x < 640 && posOutS.y >= 0 && posOutS.y < 800) {
            uint2 posOut = make_uint2(uint(posOutS.x), uint(posOutS.y));
            out.el() = viewLeft[posOut];
        }
    } else {
        int2 hmd_lens_center = make_int2(half/2 + half, h/2);
        const float2    v   = make_float2(hmd_scale_in.x * (posS.x - hmd_lens_center.x), hmd_scale_in.y * (posS.y - hmd_lens_center.y));
        const float     rr  = v.x*v.x + v.y*v.y;
        const float4    r   = make_float4(1, rr, rr*rr, rr*rr*rr);
        const float     product = hmd_warp_param.x*r.x + hmd_warp_param.y*r.y + hmd_warp_param.z*r.z + hmd_warp_param.w*r.w;
        const float2    vOut = make_float2(hmd_scale_out.x * v.x * product + hmd_lens_center.x, hmd_scale_out.y * v.y * product + hmd_lens_center.y);
        const int2      posOutS = make_int2(int(vOut.x - half), int(vOut.y));
        if (posOutS.x >= 0 && posOutS.x < 640 && posOutS.y >= 0 && posOutS.y < 800) {
            uint2 posOut = make_uint2(uint(posOutS.x), uint(posOutS.y));
            out.el() = viewRight[posOut];
        }
    }

}

__global__ void hmd_barrel(Image<uchar4> out, const Image<uchar4> viewLeft, const Image<uchar4> viewRight) {
    //float2 lensCenterLeft = make_float2(0.5f + 0.25f * 0.5f, 0.5f);
    //float2 lensCenterRight = make_float2(0.5f - 0.25f * 0.5f, 0.5f);

    float scaleFactor = 0.7f;
    float2 hmd_scale_out = make_float2((0.5/2.0f) * scaleFactor, (1.0f/2.0f) * scaleFactor * 0.5f);
    float2 hmd_scale_in = make_float2(4.0f, 4.0f);
    float4 hmd_warp_param = make_float4(1.00f, 0.22f, 0.24f, 0.00f); // these should come from the OVR SDK

    const uint2 pos = thr2pos2();
    int2 posS = make_int2(pos.x, pos.y);

    float2 hmd_lens_center = make_float2(0.5f, 0.5f);
    
    if (posS.x > 639) {
        posS.x -= 640;
        //hmd_lens_center = make_float2(0.5f - 0.25f * 0.5f, 0.5f);
    } else {
        //hmd_lens_center = make_float2(0.5f + 0.25f * 0.5f, 0.5f);
    }

    float2 posFloat = make_float2(float(posS.x)/640.0f, float(posS.y)/800.0f);

    //*/

    const float2    v   = make_float2(hmd_scale_in.x * (posFloat.x - hmd_lens_center.x), hmd_scale_in.y * (posFloat.y - hmd_lens_center.y));
    const float     rr  = v.x*v.x + v.y*v.y;
    const float4    r   = make_float4(1, rr, rr*rr, rr*rr*rr);
    const float     product = hmd_warp_param.x*r.x + hmd_warp_param.y*r.y + hmd_warp_param.z*r.z + hmd_warp_param.w*r.w;
    const float2    vOut = make_float2(hmd_scale_out.x * v.x * product + hmd_lens_center.x, hmd_scale_out.y * v.y * product + hmd_lens_center.y);

    if (vOut.x >= 0 && vOut.x < 1 && vOut.y >= 0 && vOut.y < 1) {
        uint2 posOut = make_uint2(uint(vOut.x*1280), uint(vOut.y*800));

        if (pos.x > 639)
            out.el() = viewRight[posOut];
        else
            out.el() = viewLeft[posOut];
    }

}

void renderBarrel(Image<uchar4> out, const Image<uchar4> & viewLeft, const Image<uchar4> & viewRight){
    dim3 block(32,16);
    hmd_barrel_bis<<<divup(out.size, block), block>>>(out, viewLeft, viewRight);
}

__global__ void OculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient) {
    float f = 204.4f;
    float2 iResolution = make_float2(1280, 800);
    float DistortionScale = 1.71461;
    float pp_adjust = 48.62;
    //float IPD_by2 = 0.01;
    float3 K = make_float3(1.00f, 0.22f, 0.24f);
    //mat4 iT_bh = mat4(1,0,0,0, 0,1,0,0, 0,0,1,0, 0,0,0,1);

    const uint2 pos = thr2pos2();
    float2 uv = make_float2(pos.x, pos.y);
    
    // Which eye?
    float i = uv.x <= iResolution.x/2.0 ? 0.0 : 1.0;

    // Compute Principle point
    float sx = i*2.0 - 1.0;
    float2 pp = make_float2( sx*(iResolution.x / 4.0 - pp_adjust) + iResolution.x / 2.0, iResolution.y / 2.0 );

    // Distort uv for Oculus (using res independant coords)
    float2 theta = make_float2(4.0 * (uv.x-pp.x) / iResolution.x, 4.0 * (uv.y-pp.y) / iResolution.x);
    float rSq= theta.x * theta.x + theta.y * theta.y;
    uv.x = pp.x + iResolution.x * theta.x * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);
    uv.y = pp.y + iResolution.x * theta.y * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);

    float2 uvMinusPp = make_float2(uv.x - pp.x, uv.y - pp.y);
    uvMinusPp.x /= f;
    uvMinusPp.y /= f;

    float3 ray = make_float3(uvMinusPp.x, uvMinusPp.y, 1.0f);
    float rayNorm = sqrt(ray.x*ray.x + ray.y*ray.y + ray.z*ray.z);
    ray.x /= rayNorm;
    ray.y /= rayNorm;
    ray.z /= rayNorm;
    //float3 position = make_float3(sx * IPD_by2, 0, 0);
    float3 position = make_float3(0, 0, 0);
    float3 vLeft1;
    float3 vLeft2;
    float3 vLeft3;

    Matrix4 viewLeft = view;
    Matrix4 viewRight = view;

    vLeft1 = make_float3(viewLeft.data[0].x, viewLeft.data[0].y, viewLeft.data[0].z);
    vLeft2 = make_float3(viewLeft.data[1].x, viewLeft.data[1].y, viewLeft.data[1].z);
    vLeft3 = make_float3(viewLeft.data[2].x, viewLeft.data[2].y, viewLeft.data[2].z);

    float3 vRight1;
    float3 vRight2;
    float3 vRight3;
    vRight1 = make_float3(viewRight.data[0].x, viewRight.data[0].y, viewRight.data[0].z);
    vRight2 = make_float3(viewRight.data[1].x, viewRight.data[1].y, viewRight.data[1].z);
    vRight3 = make_float3(viewRight.data[2].x, viewRight.data[2].y, viewRight.data[2].z);

    float4 hit;
    if (i == 0) {
        // ray.x = vLeft1.x * ray.x + vLeft1.y * ray.y + vLeft1.z * ray.z;
        // ray.y = vLeft2.x * ray.x + vLeft2.y * ray.y + vLeft2.z * ray.z;
        // ray.z = vLeft3.x * ray.x + vLeft3.y * ray.y + vLeft3.z * ray.z;
        hit = raycastDirPos(volume, viewLeft, nearPlane, farPlane, step, largestep, ray);
    }
    else {
        // ray.x = vRight1.x * ray.x + vRight1.y * ray.y + vRight1.z * ray.z;
        // ray.y = vRight2.x * ray.x + vRight2.y * ray.y + vRight2.z * ray.z;
        // ray.z = vRight3.x * ray.x + vRight3.y * ray.y + vRight3.z * ray.z;
        hit = raycastDirPos(volume, viewRight, nearPlane, farPlane, step, largestep, ray);
    }
    
    float3 pos3D;
    float3 normal;
    //float depth;

    if(hit.w > 0){
        pos3D = make_float3(hit);
        //depth = hit.w;
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal.x = -2;
        } else {
            normal = normalize(surfNorm);
        }
    } else {
        pos3D = make_float3(0);
        normal = make_float3(0);
        //depth = 0;
    }

    // if(normal.x == -2.0f)
    //     out.el() = make_uchar4(0,0,0,255);
    // else {
    //     const float3 diff = normalize(light - pos3D);
    //     const float dir = fmaxf(dot(normal, diff), 0.f);
    //     const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
    //     out.el() = make_uchar4(col.x, col.y, col.z, 255);
    // }

    if(normal.x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * pos3D;
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - pos3D);
        const float dir = fmaxf(dot(normal, diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderOculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> & texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    dim3 block(32,16);
    OculusCam<<<divup(out.size, block), block>>>(out, volume, texture, view, texproj, nearPlane, farPlane, step, largestep, light, ambient);
}