#include "hip/hip_runtime.h"
/*
Copyright (c) 2011-2013 Gerhard Reitmayr, TU Graz

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS
OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "kfusion.h"
#include "perfstats.h"
#include <iostream>
#include <GL/glut.h>
#include <GL/gl.h>

using namespace std;

PerfStats Stats;

__global__ void setSphere( Volume volume, const float3 center, const float radius, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float d = length(volume.pos(pos) - center);
        if(d < radius)
            volume.set(pos, make_float2(val, 0.0f));
    }
}

__global__ void setBox( Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    uint3 pos = make_uint3(thr2pos2());
    for(pos.z = 0; pos.z < volume.size.z; ++pos.z) {
        const float3 p = volume.pos(pos);
        if(min_corner.x < p.x && min_corner.y < p.y && min_corner.z < p.z && 
           p.x < max_corner.x && p.y < max_corner.y && p.z < max_corner.z )
            volume.set(pos, make_float2(val, 0.0f));
    }
}

void initVolumeWrap( Volume volume, const float val ){
    dim3 block(32,16);
    initVolume<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, make_float2(val, 0.0f));
}

void setBoxWrap(Volume volume, const float3 min_corner, const float3 max_corner, const float val ){
    dim3 block(32,16);
    setBox<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, min_corner, max_corner, val);
}

void setSphereWrap(Volume volume, const float3 center, const float radius, const float val ){
    dim3 block(32,16);
    setSphere<<<divup(dim3(volume.size.x, volume.size.y), block), block>>>(volume, center, radius, val);
}

__global__ void renderNormals( Image<uchar3> out, const Image<float3> in ){
    float3 n = in.el();
    if(n.x == -2)
        out.el() = make_uchar3(0,0,0);
    else {
        n = normalize(n);
        out.el() = make_uchar3(n.x*128 + 128, n.y*128+128, n.z*128+128);
    }
}

void renderNormalMap( Image<uchar3> out, const Image<float3> & normal ){
    dim3 block(20,20);
    renderNormals<<<divup(normal.size, block), block>>>( out, normal );
}

__global__ void renderLightKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const float3 light, const float3 ambient ){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f);
        const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
        out.el() = make_uchar4(col.x, col.y, col.z, 255);
    }
}

void renderLight( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const float3 light, const float3 ambient ){
    dim3 block(32,16);
    renderLightKernel<<<divup(out.size, block), block>>>( out, vertex, normal, light, ambient );
}

__global__ void renderTextureKernel( Image<uchar4> out, const Image<float3> vertex, const Image<float3> normal, const Image<uchar3> texture, const Matrix4 texproj, const float3 light){
    if(normal.el().x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * vertex.el();
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - vertex.el());
        const float dir = fmaxf(dot(normal.el(), diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderTexture( Image<uchar4> out, const Image<float3> & vertex, const Image<float3> & normal, const Image<uchar3> & texture, const Matrix4 & texproj, const float3 light){
    dim3 block(32,16);
    renderTextureKernel<<<divup(out.size, block), block>>>( out, vertex, normal, texture, texproj, light);
}

__global__ void renderDepth( Image<uchar3> out, const Image<float> depth, const float nearPlane, const float farPlane){
    const float d = (clamp(depth.el(), nearPlane, farPlane) - nearPlane) / (farPlane - nearPlane);
    out.el() = make_uchar3(d * 255, d * 255, d * 255);
}

void renderDepthMap( Image<uchar3> out, const Image<float> & depth, const float nearPlane, const float farPlane ){
    dim3 block(32,16);
    renderDepth<<<divup(depth.size, block), block>>>( out, depth, nearPlane, farPlane );
}

__global__ void renderTrack( Image<uchar4> out, const Image<TrackData> data ){
    const uint2 pos = thr2pos2();
    switch(data[pos].result){
    case 1: out[pos] = make_uchar4(128, 128, 128,0);  // ok
        break;
    case -1: out[pos] = make_uchar4(0, 0, 0,0);      // no input
        break;
    case -2: out[pos] = make_uchar4(255,0,0,0);        // not in image
        break;
    case -3:  out[pos] = make_uchar4(0,255,0,0);        // no correspondence
        break;
    case -4: out[pos] = make_uchar4(0,0,255,0);        // to far away
        break;
    case -5: out[pos] = make_uchar4(255,255,0,0);     // wrong normal
        break;
    }
}

void renderTrackResult( Image<uchar4> out, const Image<TrackData> & data ){
    dim3 block(32,16);
    renderTrack<<<divup(out.size, block), block>>>( out, data );
}

__global__ void raycastLight( Image<uchar4> render, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    const uint2 pos = thr2pos2();
    int2 posS = make_int2(pos.x, pos.y);
    
    float4 hit = raycast( volume, posS, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        const float3 test = make_float3(hit);
        const float3 surfNorm = volume.grad(test);
        if(length(surfNorm) > 0){
            const float3 diff = normalize(light - test);
            const float dir = fmaxf(dot(normalize(surfNorm), diff), 0.f);
            const float3 col = clamp(make_float3(dir) + ambient, 0.f, 1.f) * 255;
            render.el() = make_uchar4(col.x, col.y, col.z,0);
        } else {
            render.el() = make_uchar4(0,0,0,0);
        }
    } else {
        render.el() = make_uchar4(0,0,0,0);
    }
}

void renderVolumeLight( Image<uchar4> out, const Volume & volume, const Matrix4 view, const float nearPlane, const float farPlane, const float largestep, const float3 light, const float3 ambient ){
    dim3 block(16,16);
    raycastLight<<<divup(out.size, block), block>>>( out,  volume, view, nearPlane, farPlane, volume.dim.x/volume.size.x, largestep, light, ambient );
}

__global__ void raycastInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    const uint2 pos = thr2pos2();

    int2 transPos = make_int2(pos.x, pos.y);
    transPos.x -= (outputSize.x - 640) / 2;
    transPos.y -= (outputSize.y - 480) / 2;
    
    float4 hit = raycast( volume, transPos, view, nearPlane, farPlane, step, largestep);
    if(hit.w > 0){
        pos3D[pos] = make_float3(hit);
        depth[pos] = hit.w;
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal[pos].x = -2;
        } else {
            normal[pos] = normalize(surfNorm);
        }
    } else {
        pos3D[pos] = make_float3(0);
        normal[pos] = make_float3(0);
        depth[pos] = 0;
    }
}

void renderInput( Image<float3> pos3D, Image<float3> normal, Image<float> depth, const Volume volume, const Matrix4 view, const float nearPlane, const float farPlane, const float step, const float largestep, const int2 outputSize){
    dim3 block(32,16);
    raycastInput<<<divup(pos3D.size, block), block>>>(pos3D, normal, depth, volume, view, nearPlane, farPlane, step, largestep, outputSize);
}

__global__ void OculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient) {
    float f = 204.4f;
    float2 iResolution = make_float2(1280, 800);
    float DistortionScale = 1.71461;
    float pp_adjust = 48.62;
    //float IPD_by2 = 0.01;
    float3 K = make_float3(1.00f, 0.22f, 0.24f);
    //mat4 iT_bh = mat4(1,0,0,0, 0,1,0,0, 0,0,1,0, 0,0,0,1);

    const uint2 pos = thr2pos2();
    float2 uv = make_float2(pos.x, pos.y);
    
    // Which eye?
    float i = uv.x <= iResolution.x/2.0 ? 0.0 : 1.0;

    // Compute Principle point
    float sx = i*2.0 - 1.0;
    float2 pp = make_float2( sx*(iResolution.x / 4.0 - pp_adjust) + iResolution.x / 2.0, iResolution.y / 2.0 );

    // Distort uv for Oculus (using res independant coords)
    float2 theta = make_float2(4.0 * (uv.x-pp.x) / iResolution.x, 4.0 * (uv.y-pp.y) / iResolution.x);
    float rSq= theta.x * theta.x + theta.y * theta.y;
    uv.x = pp.x + iResolution.x * theta.x * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);
    uv.y = pp.y + iResolution.x * theta.y * (K.x + rSq*(K.y + rSq*K.z ) ) / (4.0 * DistortionScale);

    float2 uvMinusPp = make_float2(uv.x - pp.x, uv.y - pp.y);
    uvMinusPp.x /= f;
    uvMinusPp.y /= f;

    float3 ray = make_float3(uvMinusPp.x, uvMinusPp.y, 1.0f);
    float rayNorm = sqrt(ray.x*ray.x + ray.y*ray.y + ray.z*ray.z);
    ray.x /= rayNorm;
    ray.y /= rayNorm;
    ray.z /= rayNorm;

    float4 hit;
    float3 pos3D;
    float3 normal;

    hit = raycastDirPos(volume, view, nearPlane, farPlane, step, largestep, ray);

    if(hit.w > 0){
        pos3D = make_float3(hit);
        float3 surfNorm = volume.grad(make_float3(hit));
        if(length(surfNorm) == 0){
            normal.x = -2;
        } else {
            normal = normalize(surfNorm);
        }
    } else {
        pos3D = make_float3(0);
        normal = make_float3(0);
    }

    if(normal.x == -2.0f)
        out.el() = make_uchar4(0,0,0,255);
    else {
        const float3 proj = texproj * pos3D;
        const float2 projPixel = make_float2( proj.x / proj.z + 0.5f, proj.y / proj.z + 0.5f);
        
        const float3 diff = normalize(light - pos3D);
        const float dir = fmaxf(dot(normal, diff), 0.f); // * 255;
        if(projPixel.x < 0 || projPixel.x > texture.size.x-1 || projPixel.y < 0 || projPixel.y > texture.size.y-1 ){
            out.el() = make_uchar4(dir*255,dir*255,dir*255,255);
        } else {
            const uchar3 texcol = texture[make_uint2(projPixel.x, projPixel.y)];
            out.el() = make_uchar4(texcol.x*dir, texcol.y*dir, texcol.z*dir, 255);
        }
    }
}

void renderOculusCam(Image<uchar4> out, const Volume volume, const Image<uchar3> & texture, const Matrix4 view, const Matrix4 texproj, const float nearPlane, const float farPlane, const float step, const float largestep, const float3 light, const float3 ambient){
    dim3 block(32,16);
    OculusCam<<<divup(out.size, block), block>>>(out, volume, texture, view, texproj, nearPlane, farPlane, step, largestep, light, ambient);
}